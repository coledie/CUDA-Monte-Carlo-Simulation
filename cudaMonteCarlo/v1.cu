#include "hip/hip_runtime.h"
/*
// Monte carlo approach to solving the dice game craps.
//
// Rules
// -----
// Roll 2 dice and sum values.
//
// Part 1: First Roll
// * Win if roll 7 or 11
// * Loose if roll 2, 3, or 12
// * Go onto Part 2 if did not win or loose.
//
// Part 2: Roll until win/loose
// * Win if re-roll number from Part 1
// * Loose if roll 7
*/
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hipblas.h"


__global__ void roll2(int* out) {
	/*
	Roll two dice.

	hiprand_uniform, range = (0, 1]

	Stable over ????? threads.
	*/
	int threadx = threadIdx.x + (threadIdx.y * 512);

	hiprandState state;
	hiprand_init((unsigned long long)clock() + threadx, 0, 0, &state);

	int die1 = (int) ceil(hiprand_uniform(&state) * 6);
	int die2 = (int) ceil(hiprand_uniform(&state) * 6);

	out[threadx] = die1 + die2;
}


__global__ void craps_part1(int* rolls_curr, bool* game_state, bool* outcomes) {
	/*
	First roll of game of craps, update wins and losses.
	*/
	int threadx = threadIdx.x + (threadIdx.y * 512);

	if (!game_state[threadx])
		return;

	int value = rolls_curr[threadx];

	if (value == 7 || value == 11) {
		game_state[threadx] = false;
		outcomes[threadx] = true;
	} else if (value == 2 || value == 3 || value == 12) {
		game_state[threadx] = false;
	}
}


__global__ void craps_part2(int* rolls_curr, int* rolls_original, bool* game_state, bool* outcomes) {
	int threadx = threadIdx.x + (threadIdx.y * 512);

	if (!game_state[threadx])
		return;

	int value = rolls_curr[threadx];

	if (value == rolls_original[threadx]) {
		game_state[threadx] = false;
		outcomes[threadx] = true;
	}
	else if (value == 7) {
		game_state[threadx] = false;
	}
}


int main(void) {
	unsigned int BLOCKS = 1;
	unsigned int GAME_PER_BLOCK = 512;
	unsigned int N_GAMES = BLOCKS * GAME_PER_BLOCK;
	bool game_active = true;

	unsigned int wins_total = 0;
	bool* c_outcomes = new bool[N_GAMES], * c_states = new bool[N_GAMES];

	bool* all_false = new bool[N_GAMES], *all_true = new bool[N_GAMES];
	for (int i = 0; i < N_GAMES; i++) {
		all_false[i] = false;
		all_true[i] = true;
	}

	bool* game_states = nullptr;
	hipMalloc((void**)&game_states, (int)N_GAMES * sizeof(bool));
	hipMemcpy(game_states, all_true, (int)N_GAMES * sizeof(bool), hipMemcpyHostToDevice);
	bool* outcomes = nullptr;
	hipMalloc((void**)&outcomes, (int)N_GAMES * sizeof(bool));
	hipMemcpy(outcomes, all_false, (int)N_GAMES * sizeof(bool), hipMemcpyHostToDevice);

	//
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	//// Part 1
	int* rolls_part1 = nullptr;
	hipMalloc((void**)&rolls_part1, (int)N_GAMES * sizeof(int));

	roll2 << <BLOCKS, GAME_PER_BLOCK >> > (rolls_part1);
	craps_part1 << <BLOCKS, GAME_PER_BLOCK >> > (rolls_part1, game_states, outcomes);
	hipDeviceSynchronize();

	hipMemcpy(c_outcomes, outcomes, (int)N_GAMES * sizeof(bool), hipMemcpyDeviceToHost);
	for (int i = 0; i < N_GAMES; i++)
		wins_total += (int)c_outcomes[i];

	//// Part 2
	int* rolls_part2 = nullptr;
	hipMalloc((void**)&rolls_part2, (int)N_GAMES * sizeof(int));

	game_active = false;
	hipMemcpy(c_states, game_states, (int)N_GAMES * sizeof(bool), hipMemcpyDeviceToHost);
	for (int i = 0; i < N_GAMES; i++) {
		if(c_states[i]) {
			game_active = true;
			break;
		}
	}

	while (game_active) {
		hipMemcpy(outcomes, all_false, (int)N_GAMES * sizeof(bool), hipMemcpyHostToDevice);

		roll2 << <BLOCKS, GAME_PER_BLOCK >> > (rolls_part2);
		craps_part2 << <BLOCKS, GAME_PER_BLOCK >> > (rolls_part2, rolls_part1, game_states, outcomes);
		hipDeviceSynchronize();  // Adds latency

		hipMemcpy(c_outcomes, outcomes, (int)N_GAMES * sizeof(bool), hipMemcpyDeviceToHost);
		for (int i = 0; i < N_GAMES; i++)
			wins_total += (int)c_outcomes[i];
		
		game_active = false;
		hipMemcpy(c_states, game_states, (int)N_GAMES * sizeof(bool), hipMemcpyDeviceToHost);
		for (int i = 0; i < N_GAMES; i++) {
			if (c_states[i]) {
				game_active = true;
				break;
			}
		}
	}

	//
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	printf(
		"Win pct: %0.3f Time: %5.dms\n",
		wins_total / (float)N_GAMES,
		std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()
	);

	return EXIT_SUCCESS;
}
